#include <cstdio>    
#include "hip/hip_runtime.h"    

__global__ void helloWorld()
{
  printf("Hello from block %d/%d\n",
      blockIdx.x, gridDim.x);
}

int main(int argc, char **argv)
{
  helloWorld<<<6, 10>>>();
  hipDeviceSynchronize();
  return 0;
}


