#include <cstdio>
#include "hip/hip_runtime.h"

__global__ void helloWorld()
{
  printf("Hello from block %d/%d\n",
      blockIdx.x, gridDim.x);
}

int main(int argc, char **argv)
{
  helloWorld<<<6,1>>>();
  hipDeviceSynchronize();
  return 0;
}


