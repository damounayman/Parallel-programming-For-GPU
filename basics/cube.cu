#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void cube(float *deviceOut, float *deviceIn)
{
    int idx = threadIdx.x;
    float f = deviceIn[idx];
    deviceOut[idx] = f * f * f;
}

int main(void)
{

    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in[i] = float(i);
    }

    float h_out[ARRAY_SIZE];

    float *d_in;
    float *d_out;

    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    cube<<<1, ARRAY_SIZE>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int  i =0;i<ARRAY_SIZE; i++)
	{
	    printf("%f%s", h_out[i], (i % 4 != 3) ? "\t" : "\n");


	}


    hipFree(d_in);
	hipFree(d_out);
	return 0;


}